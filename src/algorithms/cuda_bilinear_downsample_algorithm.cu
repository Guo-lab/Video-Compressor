#include "hip/hip_runtime.h"
#include "algorithms/bilinear_downsample_algorithm.hpp"
#include <hip/hip_runtime.h>
#include <>

namespace vcompress {
namespace algorithm {

/**
 * @brief CUDA kernel for bilinear downsampling.
 */
__global__ void bilinearDownsampleKernel(const uint8_t *src, uint8_t *dst, int src_width, int src_height,
                                         int dst_width, int dst_height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < dst_width && y < dst_height) {
        float x_ratio = static_cast<float>(src_width - 1) / dst_width;
        float y_ratio = static_cast<float>(src_height - 1) / dst_height;

        float src_x = x * x_ratio;
        float src_y = y * y_ratio;
        int x_floor = static_cast<int>(src_x);
        int y_floor = static_cast<int>(src_y);
        int x_ceil = min(x_floor + 1, src_width - 1);
        int y_ceil = min(y_floor + 1, src_height - 1);
        float x_fraction = src_x - x_floor;
        float y_fraction = src_y - y_floor;

        for (int c = 0; c < 3; c++) {
            uint8_t p00 = src[(y_floor * src_width + x_floor) * 3 + c];
            uint8_t p01 = src[(y_floor * src_width + x_ceil) * 3 + c];
            uint8_t p10 = src[(y_ceil * src_width + x_floor) * 3 + c];
            uint8_t p11 = src[(y_ceil * src_width + x_ceil) * 3 + c];

            float top = p00 * (1.0f - x_fraction) + p01 * x_fraction;
            float bottom = p10 * (1.0f - x_fraction) + p11 * x_fraction;
            float result = top * (1.0f - y_fraction) + bottom * y_fraction;
            dst[(y * dst_width + x) * 3 + c] = static_cast<uint8_t>(result + 0.5f);
        }
    }
}

/**
 * @brief CUDA kernel for bilinear upsampling.
 */
__global__ void bilinearUpsampleKernel(const uint8_t *src, uint8_t *dst, int src_width, int src_height,
                                       int dst_width, int dst_height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < dst_width && y < dst_height) { // within destination image bounds
        float x_ratio = static_cast<float>(src_width - 1) / (dst_width - 1);
        float y_ratio = static_cast<float>(src_height - 1) / (dst_height - 1);

        float src_x = x * x_ratio;
        float src_y = y * y_ratio;
        int x_floor = static_cast<int>(src_x);
        int y_floor = static_cast<int>(src_y);
        int x_ceil = min(x_floor + 1, src_width - 1);
        int y_ceil = min(y_floor + 1, src_height - 1);
        float x_fraction = src_x - x_floor;
        float y_fraction = src_y - y_floor;

        for (int c = 0; c < 3; c++) {
            uint8_t p00 = src[(y_floor * src_width + x_floor) * 3 + c];
            uint8_t p01 = src[(y_floor * src_width + x_ceil) * 3 + c];
            uint8_t p10 = src[(y_ceil * src_width + x_floor) * 3 + c];
            uint8_t p11 = src[(y_ceil * src_width + x_ceil) * 3 + c];

            float top = p00 * (1.0f - x_fraction) + p01 * x_fraction;
            float bottom = p10 * (1.0f - x_fraction) + p11 * x_fraction;
            float result = top * (1.0f - y_fraction) + bottom * y_fraction;
            dst[(y * dst_width + x) * 3 + c] = static_cast<uint8_t>(result + 0.5f);
        }
    }
}

CudaBilinearDownsampleAlgorithm::CudaBilinearDownsampleAlgorithm() : BilinearDownsampleAlgorithm() {
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    if (error != hipSuccess || deviceCount == 0) {
        std::cerr << "WARNING: No CUDA devices found! Falling back to CPU implementation." << std::endl;
        m_cuda_available = false;
    } else {
        m_cuda_available = true;
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);
        std::cout << "Using CUDA Device: " << deviceProp.name << std::endl;
    }
}

CudaBilinearDownsampleAlgorithm::~CudaBilinearDownsampleAlgorithm() = default;

bool CudaBilinearDownsampleAlgorithm::initialize(const CompressionConfig &config) {
    bool result = BilinearDownsampleAlgorithm::initialize(config);
    if (m_cuda_available)
        std::cout << "Initialized CUDA bilinear downsample algorithm with factor: " << m_downsample_factor
                  << std::endl;
    else
        std::cout << "WARNING: CUDA not available, falling back to CPU implementation" << std::endl;
    return result;
}

/**
 * @brief Compresses a frame using CUDA bilinear downsampling when available.
 *  If CUDA is not available, it falls back to the CPU implementation.
 * @param frame The frame to compress.
 * @return The compressed frame data.
 */
std::vector<uint8_t> CudaBilinearDownsampleAlgorithm::compressFrame(const Frame &frame) {
    if (!m_cuda_available) return BilinearDownsampleAlgorithm::compressFrame(frame);

    auto start_time = std::chrono::high_resolution_clock::now();

    int original_width = frame.width;
    int original_height = frame.height;
    int target_width = original_width / m_downsample_factor;
    int target_height = original_height / m_downsample_factor;
    std::vector<uint8_t> downsampled(target_width * target_height * 3);
    cudaDownsampleBilinear(frame.data.data(), downsampled.data(), original_width, original_height,
                           target_width, target_height);

    double original_size = original_width * original_height * 3;
    double compressed_size = target_width * target_height * 3;
    double ratio = original_size / compressed_size;

    m_stats.frames_compressed++;
    m_stats.average_compression_ratio =
        ((m_stats.average_compression_ratio * (m_stats.frames_compressed - 1)) + ratio) /
        m_stats.frames_compressed;

    // Create compressed data format: | width (4) | height (4) | raw pixel data |
    std::vector<uint8_t> compressed_data(METADATA_BYTES + downsampled.size());
    std::memcpy(compressed_data.data(), &original_width, WIDTH_BYTES);
    std::memcpy(compressed_data.data() + WIDTH_BYTES, &original_height, HEIGHT_BYTES);
    std::memcpy(compressed_data.data() + METADATA_BYTES, downsampled.data(), downsampled.size());

    auto end_time = std::chrono::high_resolution_clock::now();
    double elapsed_ms = std::chrono::duration<double, std::milli>(end_time - start_time).count();
    m_stats.total_compression_time_ms += elapsed_ms;

    return compressed_data;
}

/**
 * @brief Decompresses a frame using CUDA bilinear upsampling when available.
 * If CUDA is not available, it falls back to the CPU implementation.
 * @param compressed_data The compressed data to decompress.
 * @return The decompressed frame.
 */
Frame CudaBilinearDownsampleAlgorithm::decompressFrame(const std::vector<uint8_t> &compressed_data) {
    if (!m_cuda_available) return BilinearDownsampleAlgorithm::decompressFrame(compressed_data);

    auto start_time = std::chrono::high_resolution_clock::now();

    int original_width, original_height;
    std::memcpy(&original_width, compressed_data.data(), WIDTH_BYTES);
    std::memcpy(&original_height, compressed_data.data() + WIDTH_BYTES, HEIGHT_BYTES);

    int downsampled_width = original_width / m_downsample_factor;
    int downsampled_height = original_height / m_downsample_factor;
    const uint8_t *downsampled_data = compressed_data.data() + METADATA_BYTES;
    std::vector<uint8_t> upsampled(original_width * original_height * 3);

    cudaUpsampleBilinear(downsampled_data, upsampled.data(), downsampled_width, downsampled_height,
                         original_width, original_height);

    Frame decompressed_frame(original_width, original_height);
    decompressed_frame.data = std::move(upsampled);
    decompressed_frame.type = KEY_FRAME;

    m_stats.frames_decompressed++;

    auto end_time = std::chrono::high_resolution_clock::now();
    double elapsed_ms = std::chrono::duration<double, std::milli>(end_time - start_time).count();
    m_stats.total_decompression_time_ms += elapsed_ms;

    return decompressed_frame;
}

/**
 * @brief CUDA version for bilinear downsampling.
 *  (i) Allocate device memory for source and destination images.
 *  (ii) Copy source image to device.
 *  (iii) Launch kernel for downsampling.
 *  (iv) Check for kernel errors and synchronize; copy result back to host.
 */
void CudaBilinearDownsampleAlgorithm::cudaDownsampleBilinear(const uint8_t *src, uint8_t *dst, int src_width,
                                                             int src_height, int dst_width, int dst_height) {
    uint8_t *d_src = nullptr, *d_dst = nullptr;
    auto cudaCleanup = [&]() {
        if (d_src) hipFree(d_src);
        if (d_dst) hipFree(d_dst);
    };
    size_t src_size = src_width * src_height * 3;
    size_t dst_size = dst_width * dst_height * 3;

    hipError_t error;
    if ((error = hipMalloc(&d_src, src_size)) != hipSuccess) {
        std::cerr << "Source allocation failed: " << hipGetErrorString(error) << std::endl;
        return;
    }
    if ((error = hipMalloc(&d_dst, dst_size)) != hipSuccess) {
        std::cerr << "Destination allocation failed: " << hipGetErrorString(error) << std::endl;
        cudaCleanup();
        return;
    }

    if ((error = hipMemcpy(d_src, src, src_size, hipMemcpyHostToDevice)) != hipSuccess) {
        std::cerr << "Host->device copy failed: " << hipGetErrorString(error) << std::endl;
        cudaCleanup();
        return;
    }

    // Launch downsampling kernel
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((dst_width + blockSize.x - 1) / blockSize.x, (dst_height + blockSize.y - 1) / blockSize.y);
    bilinearDownsampleKernel<<<gridSize, blockSize>>>(d_src, d_dst, src_width, src_height, dst_width,
                                                      dst_height);

    if ((error = hipGetLastError()) != hipSuccess || (error = hipDeviceSynchronize()) != hipSuccess) {
        std::cerr << "Kernel execution failed: " << hipGetErrorString(error) << std::endl;
        cudaCleanup();
        return;
    }

    if ((error = hipMemcpy(dst, d_dst, dst_size, hipMemcpyDeviceToHost)) != hipSuccess) {
        std::cerr << "Device->host copy failed: " << hipGetErrorString(error) << std::endl;
    }
    cudaCleanup();
}

/**
 * @brief CUDA version for bilinear upsampling.
 *  (i) Allocate device memory for source and destination images.
 *  (ii) Copy source image to device.
 *  (iii) Launch kernel for upsampling.
 *  (iv) Check for kernel errors and synchronize; copy result back to host.
 *  (v) Free device memory.
 */
void CudaBilinearDownsampleAlgorithm::cudaUpsampleBilinear(const uint8_t *src, uint8_t *dst, int src_width,
                                                           int src_height, int dst_width, int dst_height) {
    uint8_t *d_src = nullptr, *d_dst = nullptr;
    size_t src_size = src_width * src_height * 3;
    size_t dst_size = dst_width * dst_height * 3;
    hipError_t error;
    auto cudaCleanup = [&]() {
        if (d_src) hipFree(d_src);
        if (d_dst) hipFree(d_dst);
    };

    if ((error = hipMalloc(&d_src, src_size)) != hipSuccess) { // Allocate source buffer
        std::cerr << "Source allocation failed: " << hipGetErrorString(error) << std::endl;
        return;
    }
    if ((error = hipMalloc(&d_dst, dst_size)) != hipSuccess) { // Allocate destination buffer
        std::cerr << "Destination allocation failed: " << hipGetErrorString(error) << std::endl;
        cudaCleanup();
        return;
    }

    if ((error = hipMemcpy(d_src, src, src_size, hipMemcpyHostToDevice)) != hipSuccess) {
        std::cerr << "Host->device copy failed: " << hipGetErrorString(error) << std::endl;
        cudaCleanup();
        return;
    }

    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((dst_width + blockSize.x - 1) / blockSize.x, (dst_height + blockSize.y - 1) / blockSize.y);
    bilinearUpsampleKernel<<<gridSize, blockSize>>>(d_src, d_dst, src_width, src_height, dst_width,
                                                    dst_height);

    if ((error = hipGetLastError()) != hipSuccess ||
        (error = hipDeviceSynchronize()) != hipSuccess) { // Check for kernel errors and synchronize
        std::cerr << "Kernel execution failed: " << hipGetErrorString(error) << std::endl;
        cudaCleanup();
        return;
    }

    if ((error = hipMemcpy(dst, d_dst, dst_size, hipMemcpyDeviceToHost)) != hipSuccess) {
        std::cerr << "Device->host copy failed: " << hipGetErrorString(error) << std::endl;
    }
    cudaCleanup();
}

} // namespace algorithm
} // namespace vcompress